#include <hip/hip_runtime.h>

__global__ void softmax_kernel(const float* input, float* output, int N) {

}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    softmax_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}