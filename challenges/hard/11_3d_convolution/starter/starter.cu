#include <hip/hip_runtime.h>

// input, kernel, output are device pointers
extern "C" void solve(const float* input, const float* kernel, float* output,
           int input_depth, int input_rows, int input_cols,
           int kernel_depth, int kernel_rows, int kernel_cols) {
            
}